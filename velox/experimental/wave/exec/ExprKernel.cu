#include "hip/hip_runtime.h"
/*
 * Copyright (c) Facebook, Inc. and its affiliates.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "velox/experimental/wave/exec/ExprKernel.h"

#include <gflags/gflags.h>
#include "velox/experimental/wave/common/Block.cuh"
#include "velox/experimental/wave/common/CudaUtil.cuh"
#include "velox/experimental/wave/exec/WaveCore.cuh"
#include "velox/experimental/wave/exec/Aggregate.cuh"

DEFINE_bool(kernel_gdb, false, "Run kernels sequentially for debugging");

namespace facebook::velox::wave {

template <typename T>
__device__ inline T opFunc_kPlus(T left, T right) {
  return left + right;
}

template <typename T, typename OpFunc>
__device__ inline void binaryOpKernel(
    OpFunc func,
    IBinary& instr,
    Operand** operands,
    int32_t blockBase,
    char* shared,
    ErrorCode& laneStatus) {
  if (!laneActive(laneStatus)) {
    return;
  }
  T left;
  T right;
  if (operandOrNull(operands, instr.left, blockBase, shared, left) &&
      operandOrNull(operands, instr.right, blockBase, shared, right)) {
    flatResult<decltype(func(left, right))>(
        operands, instr.result, blockBase, shared) = func(left, right);
  } else {
    resultNull(operands, instr.result, blockBase, shared);
  }
}

__device__ void filterKernel(
    const IFilter& filter,
    Operand** operands,
    int32_t blockBase,
        WaveShared* shared,
    ErrorCode& laneStatus) {
  bool isPassed = laneActive(laneStatus);
  if (isPassed) {
    if (!operandOrNull(operands, filter.flags, blockBase, &shared->data, isPassed)) {
      isPassed = false;
    }
  }
  uint32_t bits = __ballot_sync(0xffffffff, isPassed);
  if (threadIdx.x == 0) {
    reinterpret_cast<int32_t*>(&shared->data)[threadIdx.x / kWarpThreads] = __popc(bits);
  }
  __syncthreads();
  if (threadIdx.x < kWarpThreads) {
    constexpr int32_t kNumWarps = kBlockSize / kWarpThreads;
    int32_t cnt = threadIdx.x < kNumWarps ? reinterpret_cast<int32_t*>(&shared->data)[threadIdx.x] : 0;
    int32_t sum;
    using Scan = hipcub::WarpScan<int32_t, kBlockSize / kWarpThreads>;
    Scan(*reinterpret_cast<Scan::TempStorage*>(shared)).ExclusiveSum(cnt, sum);
    if (threadIdx.x < kNumWarps) {
      if (threadIdx.x == kNumWarps - 1) {
	shared->numRows = cnt + sum;
      }
      reinterpret_cast<int32_t*>(&shared->data)[threadIdx.x] = sum;
      }
  }
  __syncthreads();
  if (bits & (1 << threadIdx.x & (kWarpThreads-1))) {
    auto* indices = reinterpret_cast<int32_t*>(operands[filter.indices]->base);
    auto start = reinterpret_cast<int32_t*>(&shared->data)[threadIdx.x / kWarpThreads];
    auto bit = start + __popc(bits & lowMask<uint32_t>(threadIdx.x & (kWarpThreads - 1)));
    indices[bit] = blockBase + threadIdx.x;
  }
  laneStatus = threadIdx.x < shared->numRows ? ErrorCode::kOk : ErrorCode::kInactive;
}

__device__ void wrapKernel(
    const IWrap& wrap,
    Operand** operands,
    int32_t blockBase,
    int32_t numRows,
    void* shared) {
  Operand* op = operands[wrap.indices];
  auto* filterIndices = reinterpret_cast<int32_t*>(op->base);
  if (filterIndices[blockBase + numRows - 1] == numRows + blockBase - 1) {
    // There is no cardinality change.
    return;
  }

  struct WrapState {
    int32_t* indices;
  };

  auto* state = reinterpret_cast<WrapState*>(shared);
  bool rowActive = threadIdx.x < numRows;
  for (auto column = 0; column < wrap.numColumns; ++column) {
    if (threadIdx.x == 0) {
      auto opIndex = wrap.columns[column];
      auto* op = operands[opIndex];
      int32_t** opIndices = &op->indices[blockBase / kBlockSize];
      if (!*opIndices) {
        *opIndices = filterIndices + blockBase;
        state->indices = nullptr;
      } else {
        state->indices = *opIndices;
      }
    }
    __syncthreads();
    // Every thread sees the decision on thred 0 above.
    if (!state->indices) {
      continue;
    }
    int32_t newIndex;
    if (rowActive) {
      newIndex =
          state->indices[filterIndices[blockBase + threadIdx.x] - blockBase];
    }
    // All threads hit this.
    __syncthreads();
    if (rowActive) {
      state->indices[threadIdx.x] = newIndex;
    }
  }
  __syncthreads();
}

#define BINARY_TYPES(opCode, OP)                             \
  case OP_MIX(opCode, WaveTypeKind::BIGINT):                 \
    binaryOpKernel<int64_t>(                                 \
        [](auto left, auto right) { return left OP right; }, \
        instruction->_.binary,                               \
        operands,                                            \
        blockBase,                                           \
        &shared->data,                                              \
        laneStatus);                                             \
    break;

__global__ void waveBaseKernel(
			       KernelParams params) {
  extern __shared__ __align__(16) char sharedChar[];
  WaveShared* shared = reinterpret_cast<WaveShared*>(sharedChar);
  int programIndex = params.programIdx[blockIdx.x];
  auto* program = params.programs[programIndex];
  if (threadIdx.x == 0) {
    shared->operands = params.operands[programIndex];
    shared->status = &params.status[blockIdx.x - params.blockBase[blockIdx.x]];
    shared->numRows = shared->status->numRows;
    shared->blockBase = (blockIdx.x - params.blockBase[blockIdx.x]) * blockDim.x;
    shared->states = params.operatorStates[programIndex];
    shared->stop = false;
  }
  __syncthreads();
  auto blockBase = shared->blockBase;
  auto operands = shared->operands;
  ErrorCode laneStatus;
    Instruction* instruction;
  if (params.startPC == nullptr) {
    instruction = program->instructions;
    laneStatus = threadIdx.x < shared->numRows ? ErrorCode::kOk : ErrorCode::kInactive;
  } else {
    instruction = program->instructions + params.startPC[programIndex];
    laneStatus = shared->status->errors[threadIdx.x];
  }
  for (;;) {
    switch (instruction->opCode) {
      case OpCode::kReturn:
	if (threadIdx.x == 0) {
	  shared->status->numRows = shared->numRows;
	}
	  shared->status->errorCode[threadIdx.x] = lanestatus;
        __syncthreads();
        return;
      case OpCode::kFilter:
        filterKernel(
            instruction->_.filter,
            operands,
            blockBase,
	    shared,
	    laneStatus);
        break;

      case OpCode::kWrap:
        wrapKernel(
            instruction->_.wrap, operands, blockBase, status->numRows, shared);
        break;
    case OpCode::kAggregate:
      aggregateKernel(instruction->_.aggregate, shared);
      break;
    case OpCode::kReadAggregate:
      readAggregateKernel(instruction->_.aggregate, shared);
      break;
      BINARY_TYPES(OpCode::kPlus, +);
        BINARY_TYPES(OpCode::kLT, <);
    }
    ++instruction;
  }
}

int32_t instructionSharedMemory(const Instruction& instruction) {
  switch (instruction.opCode) {
    case OpCode::kFilter:
      return sizeof(WaveShared) + (2 + (kBlockSize / kWarpThreads)) * sizeof(int32_t);
  default:
    return sizeof(WaveShared);
  }
}

void WaveKernelStream::call(
    Stream* alias,
    int32_t numBlocks,
    int32_t sharedSize,
    KernelParams& params) {
  waveBaseKernel<<<
      numBlocks,
      kBlockSize,
      sharedSize,
      alias ? alias->stream()->stream : stream()->stream>>>(
							    params);
  if (FLAGS_kernel_gdb) {
    (alias ? alias : this)->wait();
  }
}
  
REGISTER_KERNEL("expr", waveBaseKernel);

  void __global__ setupAggregationKernel(AggregationControl op) {
  //    assert(op.maxTableEntries == 0);
  auto* data = new(op.head) DeviceAggregation();
  data->rowSize = op.rowSize;
  data->singleRow = reinterpret_cast<char*>(data + 1);
  memset(data->singleRow, 0, op.rowSize);
}
  
  void WaveKernelStream::setupAggregation(AggregationControl& op) {
    setupAggregationKernel<<<1, 1, 0, stream_->stream>>>(op);
    wait();
  }

  
} // namespace facebook::velox::wave
