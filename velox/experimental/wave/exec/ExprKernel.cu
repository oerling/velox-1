#include "hip/hip_runtime.h"
/*
 * Copyright (c) Facebook, Inc. and its affiliates.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "velox/experimental/wave/exec/ExprKernel.h"

#include "velox/experimental/wave/common/Block.cuh"
#include "velox/experimental/wave/common/CudaUtil.cuh"
#include "velox/experimental/wave/exec/WaveCore.cuh"

namespace facebook::velox::wave {

template <typename T>
__device__ inline T opFunc_kPlus(T left, T right) {
  return left + right;
}

template <typename T, typename OpFunc>
__device__ inline void binaryOpKernel(
    OpFunc func,
    IBinary& instr,
    Operand** operands,
    int32_t blockBase,
    char* shared,
    BlockStatus* status) {
  if (threadIdx.x >= status->numRows) {
    return;
  }
  T left;
  T right;
  if (operandOrNull(operands, instr.left, blockBase, shared, left) &&
      operandOrNull(operands, instr.right, blockBase, shared, right)) {
    flatResult<T>(operands, instr.result, blockBase, shared) =
        func(left, right);
  } else {
    resultNull(operands, instr.result, blockBase, shared);
  }
}

__device__ void filterKernel(
    const IFilter& filter,
    Operand** operands,
    int32_t blockBase,
    char* shared,
    int32_t& numRows) {
  auto* flags = operands[filter.flags];
  auto* indices = operands[filter.indices];
  if (flags->nulls) {
    boolBlockToIndices<kBlockSize>(
        [&]() -> uint8_t {
          return threadIdx.x >= numRows
              ? 0
              : flatValue<uint8_t>(flags->base, blockBase) &
                  flatValue<uint8_t>(flags->nulls, blockBase);
        },
        blockBase,
        reinterpret_cast<int32_t*>(indices->base) + blockBase,
        shared,
        numRows);
  } else {
    boolBlockToIndices<kBlockSize>(
        [&]() -> uint8_t {
          return threadIdx.x >= numRows
              ? 0
              : flatValue<uint8_t>(flags->base, blockBase);
        },
        blockBase,
        reinterpret_cast<int32_t*>(indices->base) + blockBase,
        shared,
        numRows);
  }
}

__device__ void wrapKernel(
    const IWrap& wrap,
    Operand** operands,
    int32_t blockBase,
    int32_t numRows) {
  if (threadIdx.x >= numRows) {
    return;
  }
  Operand* op = operands[wrap.indices];
  auto* filterIndices = reinterpret_cast<int32_t*>(op->base);
  if (filterIndices[blockBase + numRows - 1] == numRows + blockBase - 1) {
    // There is no cardinality change.
    return;
  }
  for (auto column = 0; column < wrap.numColumns; ++column) {
    int32_t opIndex = wrap.columns[column];
    auto* op = operands[opIndex];
    int32_t newIndex;
    int32_t** opIndices = &op->indices[blockBase / kBlockSize];
    bool remap = *opIndices != nullptr;
    if (remap) {
      newIndex = (*opIndices)[filterIndices[threadIdx.x]];
    }
    __syncthreads();
    if (remap) {
      if (threadIdx.x < numRows) {
        (*opIndices)[threadIdx.x] = newIndex;
      }
    } else if (threadIdx.x == 0) {
      *opIndices = filterIndices + blockBase;
    }
  }
}

#define BINARY_TYPES(opCode, OP)                             \
  case OP_MIX(opCode, WaveTypeKind::BIGINT):                 \
    binaryOpKernel<int64_t>(                                 \
        [](auto left, auto right) { return left OP right; }, \
        instruction->_.binary,                               \
        operands,                                            \
        blockBase,                                           \
        shared,                                              \
        status);                                             \
    break;

__global__ void waveBaseKernel(
    int32_t* baseIndices,
    int32_t* programIndices,
    ThreadBlockProgram** programs,
    Operand*** programOperands,
    BlockStatus* blockStatusArray) {
  using ScanAlgorithm = hipcub::BlockScan<int, 256, hipcub::BLOCK_SCAN_RAKING>;
  extern __shared__ __align__(
      alignof(typename ScanAlgorithm::TempStorage)) char shared[];
  int programIndex = programIndices[blockIdx.x];
  auto* program = programs[programIndex];
  auto* operands = programOperands[programIndex];
  auto* status = &blockStatusArray[blockIdx.x - baseIndices[blockIdx.x]];
  int32_t blockBase = (blockIdx.x - baseIndices[blockIdx.x]) * blockDim.x;
  for (auto i = 0; i < program->numInstructions; ++i) {
    auto instruction = program->instructions[i];
    switch (instruction->opCode) {
      case OpCode::kFilter:
        filterKernel(
            instruction->_.filter,
            operands,
            blockBase,
            shared,
            status->numRows);
        break;

      case OpCode::kWrap:
        wrapKernel(instruction->_.wrap, operands, blockBase, status->numRows);
        break;

        BINARY_TYPES(OpCode::kPlus, +);
        BINARY_TYPES(OpCode::kLT, <);
    }
  }
}

void WaveKernelStream::call(
    Stream* alias,
    int32_t numBlocks,
    int32_t* bases,
    int32_t* programIdx,
    ThreadBlockProgram** programs,
    Operand*** operands,
    BlockStatus* status,
    int32_t sharedSize) {
  waveBaseKernel<<<
      numBlocks,
      kBlockSize,
      sharedSize,
      alias ? alias->stream()->stream : stream()->stream>>>(
      bases, programIdx, programs, operands, status);
}

} // namespace facebook::velox::wave
