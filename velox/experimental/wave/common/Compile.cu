#include "hip/hip_runtime.h"
/*
 * Copyright (c) Facebook, Inc. and its affiliates.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */



#include "velox/experimental/wave/common/Cuda.h"
#include "velox/experimental/wave/common/CudaUtil.cuh"
#include "velox/experimental/wave/common/Exception.h"
#include <hip/hiprtc.h>

#include <gflags/gflags.h>


DEFINE_string(wavegen_include_path, "", "path to velox/experimental/wave. Mustt contain the cuh headers for runtime compilation");

namespace facebook::velox::wave {

  void nvrtcCheck(hiprtcResult result) {
    if (result != HIPRTC_SUCCESS) {                                
      waveError(hiprtcGetErrorString(result));
    }
  }

  class CompiledModuleImpl  : public CompiledModule {
    
    ~CompiledModuleImpl() {
      hipModuleUnload(module);
    }

    
    hipModule_t module;
    std::vector<hipFunction_t> kernels;
  };
  
  std::shared_ptr<CompiledModule> CompiledModule::create(const KernelSpec& spec) {

    hiprtcProgram prog;
    hiprtcCreateProgram(&prog,
		       spec.code.c_str(),         // buffer
        "rtctest.cu",    // name
        0,             // numHeaders
        NULL,          // headers
		       NULL);         // includeNames
    for (auto& name : spec.entryPoints) {
      nvrtcCheck(hiprtcAddNameExpression(entry.c_str()));
    }
    const char *opts[] = {"--gpu-architecture=compute_80", "-g", "-G"};
    hiprtcCompileProgram(prog,     // prog
			3,        // numOptions
			opts);    // options
    

    size_t logSize;

    hiprtcGetProgramLogSize(prog, &logSize);
    char *log = new char[logSize];
    hiprtcGetProgramLog(prog, log);
    // Obtain PTX from the program.
    size_t ptxSize;
    hiprtcGetCodeSize(prog, &ptxSize);
    char *ptx = new char[ptxSize];
    hiprtcGetCode(prog, ptx);
    std::vector<std::string> loweredNames;
    for (auto& entry : spec.entryPoints) {
      const char * temp;
      nvrtcCheck(hiprtcGetLoweredName(prog, entry.c_str(), &temp));
      loweredNames.push_back(std::string(temp));
    }
 
    hiprtcDestroyProgram(&prog);

    hipDevice_t cuDevice;
    hipCtx_t context;
    getDeviceAndContext(dvice, context);
    hipModule_t module;
    hipModuleLoadDataEx(&module, ptx, 0, 0, 0);
    std::vector<hipFunction_t> funcs;
    for (auto& name : loweredNames) {
      funcs.emplace_back();
      hipModuleGetFunction(&funcs.back(), module, name.c_str());
    }
    return std::make_shared<CompiledModuleImpl>(module, std::move(funcs));
}


  CompiledModuleImpl::launch(int32_t kernelIdx, int32_t numBlocks, int32_t numThreads, int32_t shared, void* stream, void** args) {
			      
   hipModuleLaunchKernel(kernels[idx],
            numBlocks, 1, 1,   // grid dim
            numThreads, 1, 1,    // block dim
		  shared, stream,             // shared mem and stream
            args,                // arguments
            0);
  };
