#include "hip/hip_runtime.h"
/*
 * Copyright (c) Facebook, Inc. and its affiliates.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */



#include "velox/experimental/wave/common/Cuda.h"
#include "velox/experimental/wave/common/CudaUtil.cuh"
#include "velox/experimental/wave/common/Exception.h"
#include <hip/hiprtc.h>

#include <gflags/gflags.h>


DEFINE_string(wavegen_include_path, "", "path to velox/experimental/wave. Mustt contain the cuh headers for runtime compilation");

namespace facebook::velox::wave {

  void nvrtcCheck(hiprtcResult result) {
    if (result != HIPRTC_SUCCESS) {                                
      waveError(hiprtcGetErrorString(result));
    }
  }

  class CompiledModuleImpl  : public CompiledModule {
  public:
    CompiledModuleImpl(hipModule_t module, std::vector<hipFunction_t> kernels)
      : module(module), kernels(std::move(kernels)) {}
    
    ~CompiledModuleImpl() {
      hipModuleUnload(module);
    }

    void launch(int32_t kernelIdx, int32_t numBlocks, int32_t numThreads, int32_t shared, Stream* stream, void** args) override;

  private:
    hipModule_t module;
    std::vector<hipFunction_t> kernels;
  };
  
  std::shared_ptr<CompiledModule> CompiledModule::create(const KernelSpec& spec) {
    hiprtcProgram prog;
    hiprtcCreateProgram(&prog,
		       spec.code.c_str(),         // buffer
		       spec.filePath.c_str(),    // name
        0,             // numHeaders
        NULL,          // headers
		       NULL);         // includeNames
    for (auto& name : spec.entryPoints) {
      nvrtcCheck(hiprtcAddNameExpression(prog, name.c_str()));
    }
    const char *opts[] = {"--gpu-architecture=compute_80",
			  "-G",
			  "-I/usr/local/cuda-12.1/targets/x86_64-linux/include/cuda/std/detail/libcxx/include",
			  "-I/usr/local/cuda-12.1/targets/x86_64-linux/include"};
    auto compileResult = hiprtcCompileProgram(prog,     // prog
					     4,        // numOptions
			opts);    // options
    

    size_t logSize;

    hiprtcGetProgramLogSize(prog, &logSize);
    std::string log;
    log.resize(logSize);
    hiprtcGetProgramLog(prog, log.data());

    if (compileResult != HIPRTC_SUCCESS) {
      waveError(std::string("Cuda compilation error: ") + log);
    }
    // Obtain PTX from the program.
    size_t ptxSize;
    nvrtcCheck(hiprtcGetCodeSize(prog, &ptxSize));
    std::string ptx;
    ptx.resize(ptxSize);
    nvrtcCheck(hiprtcGetCode(prog, ptx.data()));
    std::vector<std::string> loweredNames;
    for (auto& entry : spec.entryPoints) {
      const char * temp;
      nvrtcCheck(hiprtcGetLoweredName(prog, entry.c_str(), &temp));
      loweredNames.push_back(std::string(temp));
    }
 
    hiprtcDestroyProgram(&prog);

    hipDevice_t device;
    hipCtx_t context;
    getDeviceAndContext(device, context);
    hipModule_t module;
    CU_CHECK(hipModuleLoadDataEx(&module, ptx.data(), 0, 0, 0));
    std::vector<hipFunction_t> funcs;
    for (auto& name : loweredNames) {
      funcs.emplace_back();
      CU_CHECK(hipModuleGetFunction(&funcs.back(), module, name.c_str()));
    }
    return std::make_shared<CompiledModuleImpl>(module, std::move(funcs));
}

  void CompiledModuleImpl::launch(int32_t kernelIdx, int32_t numBlocks, int32_t numThreads, int32_t shared, Stream* stream, void** args) {
    auto result = hipModuleLaunchKernel(kernels[kernelIdx],
            numBlocks, 1, 1,   // grid dim
            numThreads, 1, 1,    // block dim
		  shared, (hipStream_t)stream->stream()->cuStream,             // shared mem and stream
            args,                // arguments
            0);
    CU_CHECK(result);
  };

}
