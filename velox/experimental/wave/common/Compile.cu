#include "hip/hip_runtime.h"
/*
 * Copyright (c) Facebook, Inc. and its affiliates.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <fmt/format.h>
#include <gflags/gflags.h>
#include <hip/hiprtc.h>
#include "velox/experimental/wave/common/Cuda.h"
#include "velox/experimental/wave/common/CudaUtil.cuh"
#include "velox/experimental/wave/common/Exception.h"

DEFINE_string(
    wavegen_architecture,
    "compute_80",
    "--gpu-architecture flag for generated code");

namespace facebook::velox::wave {

void nvrtcCheck(hiprtcResult result) {
  if (result != HIPRTC_SUCCESS) {
    waveError(hiprtcGetErrorString(result));
  }
}

class CompiledModuleImpl : public CompiledModule {
 public:
  CompiledModuleImpl(hipModule_t module, std::vector<hipFunction_t> kernels)
      : module_(module), kernels_(std::move(kernels)) {}

  ~CompiledModuleImpl() {
    auto result = hipModuleUnload(module_);
    if (result != hipSuccess) {
      LOG(ERROR) << "Error in unloading module " << result;
    }
  }

  void launch(
      int32_t kernelIdx,
      int32_t numBlocks,
      int32_t numThreads,
      int32_t shared,
      Stream* stream,
      void** args) override;

  KernelInfo info(int32_t kernelIdx) override;

 private:
  hipModule_t module_;
  std::vector<hipFunction_t> kernels_;
};

std::shared_ptr<CompiledModule> CompiledModule::create(const KernelSpec& spec) {
  hiprtcProgram prog;
  hiprtcCreateProgram(
      &prog,
      spec.code.c_str(), // buffer
      spec.filePath.c_str(), // name
      spec.numHeaders, // numHeaders
      spec.headers, // headers
      spec.headerNames); // includeNames
  for (auto& name : spec.entryPoints) {
    nvrtcCheck(hiprtcAddNameExpression(prog, name.c_str()));
  }
  auto architecture =
      fmt::format("--gpu-architecture={}", FLAGS_wavegen_architecture);
  const char* opts[] = {
      architecture.c_str(),
#ifndef NDEBUG
      "-G"
      #else
      "-O3"
#endif
  };
  auto compileResult = hiprtcCompileProgram(
      prog, // prog
      sizeof(opts) / sizeof(char*), // numOptions
      opts); // options

  size_t logSize;

  hiprtcGetProgramLogSize(prog, &logSize);
  std::string log;
  log.resize(logSize);
  hiprtcGetProgramLog(prog, log.data());

  if (compileResult != HIPRTC_SUCCESS) {
    hiprtcDestroyProgram(&prog);
    waveError(std::string("Cuda compilation error: ") + log);
  }
  // Obtain PTX from the program.
  size_t ptxSize;
  nvrtcCheck(hiprtcGetCodeSize(prog, &ptxSize));
  std::string ptx;
  ptx.resize(ptxSize);
  nvrtcCheck(hiprtcGetCode(prog, ptx.data()));
  std::vector<std::string> loweredNames;
  for (auto& entry : spec.entryPoints) {
    const char* temp;
    nvrtcCheck(hiprtcGetLoweredName(prog, entry.c_str(), &temp));
    loweredNames.push_back(std::string(temp));
  }

  hiprtcDestroyProgram(&prog);

  hipModule_t module;
  CU_CHECK(hipModuleLoadDataEx(&module, ptx.data(), 0, 0, 0));
  std::vector<hipFunction_t> funcs;
  for (auto& name : loweredNames) {
    funcs.emplace_back();
    CU_CHECK(hipModuleGetFunction(&funcs.back(), module, name.c_str()));
  }
  return std::make_shared<CompiledModuleImpl>(module, std::move(funcs));
}

void CompiledModuleImpl::launch(
    int32_t kernelIdx,
    int32_t numBlocks,
    int32_t numThreads,
    int32_t shared,
    Stream* stream,
    void** args) {
  auto result = hipModuleLaunchKernel(
      kernels_[kernelIdx],
      numBlocks,
      1,
      1, // grid dim
      numThreads,
      1,
      1, // block dim
      shared,
      reinterpret_cast<hipStream_t>(stream->stream()->stream),
      args,
      0);
  CU_CHECK(result);
};

KernelInfo CompiledModuleImpl::info(int32_t kernelIdx) {
  KernelInfo info;
  auto f = kernels_[kernelIdx];
  hipFuncGetAttribute(&info.numRegs, HIP_FUNC_ATTRIBUTE_NUM_REGS, f);
  hipFuncGetAttribute(
      &info.sharedMemory, HIP_FUNC_ATTRIBUTE_SHARED_SIZE_BYTES, f);
  hipFuncGetAttribute(
      &info.maxThreadsPerBlock, HIP_FUNC_ATTRIBUTE_MAX_THREADS_PER_BLOCK, f);
  int32_t max;
  hipModuleOccupancyMaxActiveBlocksPerMultiprocessor(&max, f, 256, 0);
  info.maxOccupancy0 = max;
  hipModuleOccupancyMaxActiveBlocksPerMultiprocessor(&max, f, 256, 256 * 32);
  info.maxOccupancy32 = max;
  return info;
}

} // namespace facebook::velox::wave
