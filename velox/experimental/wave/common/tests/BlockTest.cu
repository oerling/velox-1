#include "hip/hip_runtime.h"
#include "velox/experimental/wave/common/Block.cuh"
#include "velox/experimental/wave/common/CudaUtil.cuh"
#include "velox/experimental/wave/common/tests/BlockTest.h"

namespace facebook::velox::wave {

using ScanAlgorithm = hipcub::BlockScan<int, 256, hipcub::BLOCK_SCAN_RAKING>;

__global__ void boolToIndices(
    uint8_t** bools,
    int32_t** indices,
    int32_t* sizes,
    int64_t* times) {
  extern __shared__ __align__(alignof(ScanAlgorithm::TempStorage)) char smem[];
  int32_t idx = blockIdx.x;
  // Start cycle timer
  clock_t start = clock();
  uint8_t* blockBools = bools[idx];
  boolBlockToIndices<256>(
      [&]() { return blockBools[threadIdx.x]; },
      idx * 256,
      indices[idx],
      smem,
      sizes[idx]);
  clock_t stop = clock();
  if (threadIdx.x == 0) {
    times[idx] = (start > stop) ? start - stop : stop - start;
  }
}

void BlockTestStream::testBoolToIndices(
    int32_t numBlocks,
    uint8_t** flags,
    int32_t** indices,
    int32_t* sizes,
    int64_t* times) {
  CUDA_CHECK(hipGetLastError());
  auto tempBytes = sizeof(typename ScanAlgorithm::TempStorage);
  boolToIndices<<<numBlocks, 256, tempBytes, stream_->stream>>>(
      flags, indices, sizes, times);
  CUDA_CHECK(hipGetLastError());
}

void BlockTestStream::testBoolToIndicesNoShared(
    int32_t numBlocks,
    uint8_t** flags,
    int32_t** indices,
    int32_t* sizes,
    int64_t* times,
    void* temp) {
  CUDA_CHECK(hipGetLastError());
  auto tempBytes = reinterpret_cast<char*>(temp) + blockIdx.x * sizeof(typename ScanAlgorithm::TempStorage);
  boolToIndicesNoShared<<<numBlocks, 256, tempBytes, stream_->stream>>>(
								flags, indices, sizes, times,  temp->as<void>());
  CUDA_CHECK(hipGetLastError());
}

  
__global__ void sum64(int64_t* numbers, int64_t* results) {
  extern __shared__ __align__(
      alignof(hipcub::BlockReduce<int64_t, 256>::TempStorage)) char smem[];
  int32_t idx = blockIdx.x;
  blockSum<256>(
      [&]() { return numbers[idx * 256 + threadIdx.x]; }, smem, results);
}

void BlockTestStream::testSum64(
    int32_t numBlocks,
    int64_t* numbers,
    int64_t* results) {
  auto tempBytes = sizeof(typename hipcub::BlockReduce<int64_t, 256>::TempStorage);
  sum64<<<numBlocks, 256, tempBytes, stream_->stream>>>(numbers, results);
  CUDA_CHECK(hipGetLastError());
}

  
  /// Keys and values are n sections of 8K items. The items in each section get sorted on the key.
  void   __global__ __launch_bounds__(1024) testSort(uint16_t** keys, uint16_t** values) {
    extern __shared__ __align__(16) char smem[];
    auto keyBase = keys[blockIdx.x];
    auto valueBase = values[blockIdx.x];
    blockSort([&](auto i) {
		return keyBase[i];
	      },
    [&](auto i) { return valueBase[i];},
    keys, values);
  }

  BlockStream::testSort16(int32_t numBlocks, uint16_t* keys, uint16_t* values) {
    auto tempBytes = sizeof(typename RadixSortStorage<1024, 8, uint16_t, uint16_t>);

    testSort<<<1024, numBlocks, tempBytes, stream_->stream>>>>(keys, values);
  }

} // namespace facebook::velox::wave
