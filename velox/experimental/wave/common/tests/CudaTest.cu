#include "hip/hip_runtime.h"
/*
 * Copyright (c) Facebook, Inc. and its affiliates.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "velox/experimental/wave/common/Block.cuh"
#include "velox/experimental/wave/common/CudaUtil.cuh"
#include "velox/experimental/wave/common/tests/CudaTest.h"

namespace facebook::velox::wave {
constexpr uint32_t kPrime32 = 1815531889;

  struct ResultPair {
    int64_t n;
    bool f;
  };
  
  typedef ResultPair(*TestFunc)(int64_t data, int64_t data2, bool& flag, int32_t* ptr);

__device__ TestFunc testFuncs[2];

  __device__ ResultPair testFunc(int64_t data, int64_t data2, bool& flag, int32_t* ptr){
    return {data + (data2 & 31), false};
}

void   __global__ setupFuncs() {
    testFuncs[0] = testFunc;
    testFuncs[1] = testFunc;
  }


__global__ void
incOneKernel(int32_t* numbers, int32_t size, int32_t stride, int32_t repeats) {
  for (auto counter = 0; counter < repeats; ++counter) {
    for (auto index = blockDim.x * blockIdx.x + threadIdx.x; index < size;
         index += stride) {
      ++numbers[index];
    }
    __syncthreads();
  }
}
  
__global__ void
addOneKernel(int32_t* numbers, int32_t size, int32_t stride, int32_t repeats) {
  for (auto counter = 0; counter < repeats; ++counter) {
    for (auto index = blockDim.x * blockIdx.x + threadIdx.x; index < size;
         index += stride) {
      numbers[index] += index & 31;
    }
    __syncthreads();
  }
}

__global__ void addOneSharedKernel(
    int32_t* numbers,
    int32_t size,
    int32_t stride,
    int32_t repeats) {
  extern __shared__ __align__(16) char smem[];
  int32_t* temp = reinterpret_cast<int32_t*>(smem);
  for (auto index = blockDim.x * blockIdx.x + threadIdx.x; index < size;
       index += stride) {
    temp[threadIdx.x] = numbers[index];
    for (auto counter = 0; counter < repeats; ++counter) {
      temp[threadIdx.x] += (index + counter) & 31;
    }
    __syncthreads();
    numbers[index] = temp[threadIdx.x];
  }
}

__global__ void addOneRegKernel(
    int32_t* numbers,
    int32_t size,
    int32_t stride,
    int32_t repeats) {
  for (auto index = blockDim.x * blockIdx.x + threadIdx.x; index < size;
       index += stride) {
    auto temp = numbers[index];
    for (auto counter = 0; counter < repeats; ++counter) {
      temp += (index + counter) & 31;
    }
    __syncthreads();
    numbers[index] = temp;
  }
}

  
__global__ void addOneFuncKernel(
    int32_t* numbers,
    int32_t size,
    int32_t stride,
    int32_t repeats) {
  for (auto index = blockDim.x * blockIdx.x + threadIdx.x; index < size;
       index += stride) {
    int32_t* ptr = nullptr;
    bool flag;
    auto temp = numbers[index];
    for (auto counter = 0; counter < repeats; ++counter) {
      auto result = testFuncs[counter & 1](temp, counter, flag, ptr);
      temp = result.n;
    }
    __syncthreads();
    numbers[index] = temp;
  }
}

#define TCASE(nn, m) \
        case nn: \
	temp = m + testFunc(temp, counter, flag, ptr).n; \
 break; \

__global__ void addOneSwitchKernel(
    int32_t* numbers,
    int32_t size,
    int32_t stride,
    int32_t repeats) {
  for (auto index = blockDim.x * blockIdx.x + threadIdx.x; index < size;
       index += stride) {
    int32_t* ptr = nullptr;
    bool flag;
    auto temp = numbers[index];
    for (auto counter = 0; counter < repeats; ++counter) {
      switch (counter & 31) {
	TCASE(0, 1);
	TCASE(1, 82);
	TCASE(2, 91);
	TCASE(3, 181);
	TCASE(4, 28);
	TCASE(5, 36);
	TCASE(6, 18);
	TCASE(7, 13);
	TCASE(8, 21);
	TCASE(9, 32);
	TCASE(10, 31);
	TCASE(11, 191);
	TCASE(12, 181);
	TCASE(13, 151);
	TCASE(14, 121);
	TCASE(15, 111);
	TCASE(16, 1);
	TCASE(17, 82);
	TCASE(18, 91);
	TCASE(19, 181);
	TCASE(20, 28);
	TCASE(21, 36);
	TCASE(22, 18);
	TCASE(23, 13);
	TCASE(24, 21);
	TCASE(25, 32);
	TCASE(26, 31);
	TCASE(27, 191);
	TCASE(28, 181);
	TCASE(29, 151);
	TCASE(30, 121);
	TCASE(31, 111);
    }
      }
    __syncthreads();
    numbers[index] = temp;
  }
}

#define BTCASE(nn, m) \
        asm volatile("BLK"  nn  ":"); \
	temp = m + testFunc(temp, counter, flag, ptr).n; \
	if (repeats < 1000000000) goto end;		 \



__global__ void addOneBranchKernel(
    int32_t* numbers,
    int32_t size,
    int32_t stride,
    int32_t repeats) {
  for (auto index = blockDim.x * blockIdx.x + threadIdx.x; index < size;
       index += stride) {
    int32_t* ptr = nullptr;
    bool flag;
    auto temp = numbers[index];

#pragma unroll 1
    for (auto counter = 0; counter < repeats; ++counter) {
#if 0
      if (repeats > 100000000) {
      switch (repeats) {
      case 100000001: asm volatile("bra BLK1;");
      case 100000002: asm volatile("bra BLK2;");
	break;
      }
    }
      #endif
      uint32_t bits = counter & 31;
      asm volatile("ts: .branchtargets BLK0, BLK1, BLK2, BLK3, BLK4, BLK5, BLK6, BLK7, BLK8, BLK9, BLK10, BLK11, BLK12, BLK13, BLK14, BLK15, BLK16, BLK17, BLK18, BLK19, BLK20, BLK21, BLK22, BLK23, BLK24, BLK25, BLK26, BLK27, BLK28, BLK29, BLK30, BLK31;");
      asm volatile("brx.idx %0, ts;" :: "r"(bits));


	BTCASE("0", 1);
	BTCASE("1", 82);
	BTCASE("2", 91);
	BTCASE("3", 181);
	BTCASE("4", 28);
	BTCASE("5", 36);
	BTCASE("6", 18);
	BTCASE("7", 13);
	BTCASE("8", 21);
	BTCASE("9", 32);
	BTCASE("10", 31);
	BTCASE("11", 191);
	BTCASE("12", 181);
	BTCASE("13", 151);
	BTCASE("14", 121);
	BTCASE("15", 111);
	BTCASE("16", 1);
	BTCASE("17", 82);
	BTCASE("18", 91);
	BTCASE("19", 181);
	BTCASE("20", 28);
	BTCASE("21", 36);
	BTCASE("22", 18);
	BTCASE("23", 13);
	BTCASE("24", 21);
	BTCASE("25", 32);
	BTCASE("26", 31);
	BTCASE("27", 191);
	BTCASE("28", 181);
	BTCASE("29", 151);
	BTCASE("30", 121);
	BTCASE("31", 111);
    end: ;

    }

    __syncthreads();
    numbers[index] = temp;
  }
}

__global__ void addOneFuncStoreKernel(
    int32_t* numbers,
    int32_t size,
    int32_t stride,
    int32_t repeats) {
  for (auto counter = 0; counter < repeats; ++counter) {
    for (auto index = blockDim.x * blockIdx.x + threadIdx.x; index < size;
	 index += stride) {
      int32_t* ptr = nullptr;
      bool flag;
      auto temp = numbers[index];
      numbers[index] = testFuncs[counter & 1](temp, counter, flag, ptr).n;
    }
    __syncthreads();  }
    }

void TestStream::incOne(
    int32_t* numbers,
    int32_t size,
    int32_t repeats,
    int32_t width) {
  constexpr int32_t kBlockSize = 256;
  auto numBlocks = roundUp(size, kBlockSize) / kBlockSize;
  int32_t stride = size;
  if (numBlocks > width / kBlockSize) {
    stride = width;
    numBlocks = width / kBlockSize;
  }
  incOneKernel<<<numBlocks, kBlockSize, 0, stream_->stream>>>(
      numbers, size, stride, repeats);
  CUDA_CHECK(hipGetLastError());
}

void TestStream::addOne(
    int32_t* numbers,
    int32_t size,
    int32_t repeats,
    int32_t width) {
  constexpr int32_t kBlockSize = 256;
  auto numBlocks = roundUp(size, kBlockSize) / kBlockSize;
  int32_t stride = size;
  if (numBlocks > width / kBlockSize) {
    stride = width;
    numBlocks = width / kBlockSize;
  }
  addOneKernel<<<numBlocks, kBlockSize, 0, stream_->stream>>>(
      numbers, size, stride, repeats);
  CUDA_CHECK(hipGetLastError());
}

void TestStream::addOneReg(
    int32_t* numbers,
    int32_t size,
    int32_t repeats,
    int32_t width) {
  constexpr int32_t kBlockSize = 256;
  auto numBlocks = roundUp(size, kBlockSize) / kBlockSize;
  int32_t stride = size;
  if (numBlocks > width / kBlockSize) {
    stride = width;
    numBlocks = width / kBlockSize;
  }
  addOneRegKernel<<<numBlocks, kBlockSize, 0, stream_->stream>>>(
      numbers, size, stride, repeats);
  CUDA_CHECK(hipGetLastError());
}

void TestStream::addOneFunc(
    int32_t* numbers,
    int32_t size,
    int32_t repeats,
    int32_t width) {
  constexpr int32_t kBlockSize = 256;
  setupFuncs<<<1, 1, 0, stream_->stream>>>();
  CUDA_CHECK(hipGetLastError());
  auto numBlocks = roundUp(size, kBlockSize) / kBlockSize;
  int32_t stride = size;
  if (numBlocks > width / kBlockSize) {
    stride = width;
    numBlocks = width / kBlockSize;
  }
  addOneFuncKernel<<<numBlocks, kBlockSize, 0, stream_->stream>>>(
      numbers, size, stride, repeats);
  CUDA_CHECK(hipGetLastError());
}


void TestStream::addOneFuncStore(
    int32_t* numbers,
    int32_t size,
    int32_t repeats,
    int32_t width) {
  constexpr int32_t kBlockSize = 256;
  setupFuncs<<<1, 1, 0, stream_->stream>>>();
  CUDA_CHECK(hipGetLastError());
  auto numBlocks = roundUp(size, kBlockSize) / kBlockSize;
  int32_t stride = size;
  if (numBlocks > width / kBlockSize) {
    stride = width;
    numBlocks = width / kBlockSize;
  }
  addOneFuncStoreKernel<<<numBlocks, kBlockSize, 0, stream_->stream>>>(
      numbers, size, stride, repeats);
  CUDA_CHECK(hipGetLastError());
}

void TestStream::addOneBranch(
    int32_t* numbers,
    int32_t size,
    int32_t repeats,
    int32_t width) {
  constexpr int32_t kBlockSize = 256;
  setupFuncs<<<1, 1, 0, stream_->stream>>>();
  CUDA_CHECK(hipGetLastError());
  auto numBlocks = roundUp(size, kBlockSize) / kBlockSize;
  int32_t stride = size;
  if (numBlocks > width / kBlockSize) {
    stride = width;
    numBlocks = width / kBlockSize;
  }
  addOneBranchKernel<<<numBlocks, kBlockSize, 0, stream_->stream>>>(
      numbers, size, stride, repeats);
  CUDA_CHECK(hipGetLastError());
}

void TestStream::addOneSwitch(
    int32_t* numbers,
    int32_t size,
    int32_t repeats,
    int32_t width) {
  constexpr int32_t kBlockSize = 256;
  auto numBlocks = roundUp(size, kBlockSize) / kBlockSize;
  int32_t stride = size;
  if (numBlocks > width / kBlockSize) {
    stride = width;
    numBlocks = width / kBlockSize;
  }
  addOneSwitchKernel<<<numBlocks, kBlockSize, 0, stream_->stream>>>(
      numbers, size, stride, repeats);
  CUDA_CHECK(hipGetLastError());
}
  
void TestStream::addOneShared(
    int32_t* numbers,
    int32_t size,
    int32_t repeats,
    int32_t width) {
  constexpr int32_t kBlockSize = 256;
  auto numBlocks = roundUp(size, kBlockSize) / kBlockSize;
  int32_t stride = size;
  if (numBlocks > width / kBlockSize) {
    stride = width;
    numBlocks = width / kBlockSize;
  }
  addOneSharedKernel<<<
      numBlocks,
      kBlockSize,
	kBlockSize * sizeof(int32_t),
      stream_->stream>>>(numbers, size, stride, repeats);
  CUDA_CHECK(hipGetLastError());
}

__global__ void addOneWideKernel(WideParams params) {
  auto numbers = params.numbers;
  auto size = params.size;
  auto repeat = params.repeat;
  auto stride = params.stride;
  for (auto counter = 0; counter < repeat; ++counter) {
    for (auto index = blockDim.x * blockIdx.x + threadIdx.x; index < size;
         index += stride) {
      ++numbers[index];
    }
  }
}

void TestStream::addOneWide(
    int32_t* numbers,
    int32_t size,
    int32_t repeat,
    int32_t width) {
  constexpr int32_t kBlockSize = 256;
  auto numBlocks = roundUp(size, kBlockSize) / kBlockSize;
  int32_t stride = size;
  if (numBlocks > width / kBlockSize) {
    stride = width;
    numBlocks = width / kBlockSize;
  }
  WideParams params;
  params.numbers = numbers;
  params.size = size;
  params.stride = stride;
  params.repeat = repeat;
  addOneWideKernel<<<numBlocks, kBlockSize, 0, stream_->stream>>>(params);
  CUDA_CHECK(hipGetLastError());
}

__global__ void __launch_bounds__(1024) addOneRandomKernel(
    int32_t* numbers,
    const int32_t* lookup,
    uint32_t size,
    int32_t stride,
    int32_t repeats,
    int32_t numLocal,
    int32_t localStride,
    bool emptyWarps,
    bool emptyThreads) {
  for (uint32_t counter = 0; counter < repeats; ++counter) {
    if (emptyWarps) {
      if (((threadIdx.x / 32) & 1) == 0) {
        for (auto index = blockDim.x * blockIdx.x + threadIdx.x; index < size;
             index += stride) {
          auto rnd = deviceScale32(index * (counter + 1) * kPrime32, size);
          auto sum = lookup[rnd];
          auto limit = min(rnd + localStride * (1 + numLocal), size);
          for (auto j = rnd + localStride; j < limit; j += localStride) {
            sum += lookup[j];
          }
          numbers[index] += sum;

          rnd = deviceScale32((index + 32) * (counter + 1) * kPrime32, size);
          sum = lookup[rnd];
          limit = min(rnd + localStride * (1 + numLocal), size);
          for (auto j = rnd + localStride; j < limit; j += localStride) {
            sum += lookup[j];
          }
          numbers[index + 32] += sum;
        }
      }
    } else if (emptyThreads) {
      if ((threadIdx.x & 1) == 0) {
        for (auto index = blockDim.x * blockIdx.x + threadIdx.x; index < size;
             index += stride) {
          auto rnd = deviceScale32(index * (counter + 1) * kPrime32, size);
          auto sum = lookup[rnd];
          auto limit = min(rnd + localStride * (1 + numLocal), size);
          for (auto j = rnd + localStride; j < limit; j += localStride) {
            sum += lookup[j];
          }
          numbers[index] += sum;

          rnd = deviceScale32((index + 1) * (counter + 1) * kPrime32, size);
          sum = lookup[rnd];
          limit = min(rnd + localStride * (1 + numLocal), size);
          for (auto j = rnd + localStride; j < limit; j += localStride) {
            sum += lookup[j];
          }
          numbers[index + 1] += sum;
        }
      }
    } else {
      for (auto index = blockDim.x * blockIdx.x + threadIdx.x; index < size;
           index += stride) {
        auto rnd = deviceScale32(index * (counter + 1) * kPrime32, size);
        auto sum = lookup[rnd];
        auto limit = min(rnd + localStride * (1 + numLocal), size);
        for (auto j = rnd + localStride; j < limit; j += localStride) {
          sum += lookup[j];
        }
        numbers[index] += sum;
      }
    }
    __syncthreads();
  }
  __syncthreads();
}

__global__ void __launch_bounds__(1024) addOneRandomPrefetchKernel(
    int32_t* numbers,
    const int32_t* lookup,
    uint32_t size,
    int32_t stride,
    int32_t repeats,
    int32_t numLocal,
    int32_t localStride,
    bool emptyWarps,
    bool emptyThreads) {
  for (uint32_t counter = 0; counter < repeats; ++counter) {
    if (emptyWarps) {
      if (((threadIdx.x / 32) & 1) == 0) {
        for (auto index = blockDim.x * blockIdx.x + threadIdx.x; index < size;
             index += stride) {
          auto rnd = deviceScale32(index * (counter + 1) * kPrime32, size);
          auto sum = lookup[rnd];
          auto limit = min(rnd + localStride * (1 + numLocal), size);
          for (auto j = rnd + localStride; j < limit; j += localStride) {
            sum += lookup[j];
          }
          numbers[index] += sum;

          rnd = deviceScale32((index + 32) * (counter + 1) * kPrime32, size);
          sum = lookup[rnd];
          limit = min(rnd + localStride * (1 + numLocal), size);
          for (auto j = rnd + localStride; j < limit; j += localStride) {
            sum += lookup[j];
          }
          numbers[index + 32] += sum;
        }
      }
    } else if (emptyThreads) {
      if ((threadIdx.x & 1) == 0) {
        for (auto index = blockDim.x * blockIdx.x + threadIdx.x; index < size;
             index += stride) {
          auto rnd = deviceScale32(index * (counter + 1) * kPrime32, size);
          auto sum = lookup[rnd];
          auto limit = min(rnd + localStride * (1 + numLocal), size);
          for (auto j = rnd + localStride; j < limit; j += localStride) {
            sum += lookup[j];
          }
          numbers[index] += sum;

          rnd = deviceScale32((index + 1) * (counter + 1) * kPrime32, size);
          sum = lookup[rnd];
          limit = min(rnd + localStride * (1 + numLocal), size);
          for (auto j = rnd + localStride; j < limit; j += localStride) {
            sum += lookup[j];
          }
          numbers[index + 1] += sum;
        }
      }
    } else {
      for (auto index = blockDim.x * blockIdx.x + threadIdx.x; index < size;
           index += stride) {
        auto rnd = deviceScale32(index * (counter + 1) * kPrime32, size);
        auto sum = lookup[rnd];
        auto limit = min(rnd + localStride * (1 + numLocal), size);
        for (auto j = rnd + localStride; j < limit; j += localStride) {
          sum += lookup[j];
        }
        numbers[index] += sum;
      }
    }
    __syncthreads();
  }
  __syncthreads();
}

  
void TestStream::addOneRandom(
    int32_t* numbers,
    const int32_t* lookup,
    int32_t size,
    int32_t repeats,
    int32_t width,
    int32_t numLocal,
    int32_t localStride,
    bool emptyWarps,
    bool emptyThreads) {
  constexpr int32_t kBlockSize = 256;
  auto numBlocks = roundUp(size, kBlockSize) / kBlockSize;
  int32_t stride = size;
  if (numBlocks > width / kBlockSize) {
    stride = width;
    numBlocks = width / kBlockSize;
  }
  addOneRandomKernel<<<numBlocks, kBlockSize, 0, stream_->stream>>>(
      numbers,
      lookup,
      size,
      stride,
      repeats,
      numLocal,
      localStride,
      emptyWarps,
      emptyThreads);
  CUDA_CHECK(hipGetLastError());
}

  /// Memory width and stride
  addOne64Kernel(int64_t* numbers, int32_t size, int32_t stride, int32_t repeats) {
  for (auto counter = 0; counter < repeats; ++counter) {
    for (auto index = blockDim.x * blockIdx.x + threadIdx.x; index < size;
         index += stride) {
      numbers[index] += index & 31;
    }
    __syncthreads();
  }
}

  addOne64x4ConsecKernel(int64_t* numbers, int32_t size, int32_t stride, int32_t repeats) {
    
    for (auto counter = 0; counter < repeats; ++counter) {
    for (auto index = blockDim.x * blockIdx.x + threadIdx.x; index < size;
         index += stride) {
      numbers[index] += index & 31;
    }
    __syncthreads();
  }
}

  

#define ADD8ENTRY(name, kname

  

  
REGISTER_KERNEL("addOne", addOneKernel);
REGISTER_KERNEL("addOneFunc", addOneFuncKernel);
REGISTER_KERNEL("addOneWide", addOneWideKernel);
REGISTER_KERNEL("addOneRandom", addOneRandomKernel);

} // namespace facebook::velox::wave
